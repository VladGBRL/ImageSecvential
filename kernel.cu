#include "hip/hip_runtime.h"
﻿#define _CRT_SECURE_NO_WARNINGS
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;
using namespace std::chrono;

#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        cerr << "CUDA Error: " << hipGetErrorString(code) << " " << file << " " << line << endl;
        if (abort) exit(code);
    }
}

#define TILE_WIDTH 32

__global__ void resizeKernel(
    unsigned char* input, unsigned char* output,
    int width, int height, int new_width, int new_height, int channels,
    float x_ratio, float y_ratio)
{
    int x_out = blockIdx.x * blockDim.x + threadIdx.x;
    int y_out = blockIdx.y * blockDim.y + threadIdx.y;

    if (x_out >= new_width || y_out >= new_height) return;

    int px = min(int(x_out * x_ratio), width - 1);
    int py = min(int(y_out * y_ratio), height - 1);

    int out_idx = (y_out * new_width + x_out) * channels;
    int in_idx = (py * width + px) * channels;

    for (int c = 0; c < channels; c++) {
        output[out_idx + c] = input[in_idx + c];
    }
}

int main() {
    int width, height, channels;

    auto start = high_resolution_clock::now();

    unsigned char* img = stbi_load("chemistry.jpg", &width, &height, &channels, 0);
    if (!img) {
        cerr << "Eroare la incarcarea imaginii!\n";
        return -1;
    }

    int new_width = 5000;
    int new_height = 5000;

    size_t input_size = static_cast<size_t>(width) * height * channels;
    size_t output_size = static_cast<size_t>(new_width) * new_height * channels;

    unsigned char* d_input = nullptr;
    unsigned char* d_output = nullptr;

    CUDA_CHECK(hipMalloc(&d_input, input_size));
    CUDA_CHECK(hipMalloc(&d_output, output_size));
    CUDA_CHECK(hipMemcpy(d_input, img, input_size, hipMemcpyHostToDevice));

    float x_ratio = float(width) / new_width;
    float y_ratio = float(height) / new_height;

    dim3 blockSize(TILE_WIDTH, TILE_WIDTH);
    dim3 gridSize((new_width + TILE_WIDTH - 1) / TILE_WIDTH, (new_height + TILE_WIDTH - 1) / TILE_WIDTH);

    resizeKernel << <gridSize, blockSize >> > (
        d_input, d_output,
        width, height, new_width, new_height, channels, x_ratio, y_ratio);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    unsigned char* resized_img = new unsigned char[output_size];
    CUDA_CHECK(hipMemcpy(resized_img, d_output, output_size, hipMemcpyDeviceToHost));

    stbi_write_jpg("output5.jpg", new_width, new_height, channels, resized_img, 100);

    stbi_image_free(img);
    delete[] resized_img;

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));

    auto end = high_resolution_clock::now();
    duration<double> elapsed = end - start;

    cout << "Imaginea a fost redimensionata si salvata!\n";
    cout << "Timpul de redimensionare (cu CUDA): " << elapsed.count() << " secunde\n";

    size_t free_mem, total_mem;
    CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));
    cout << "Memorie libera: " << free_mem / (1024 * 1024) << " MB din " << total_mem / (1024 * 1024) << " MB\n";

    return 0;
}
